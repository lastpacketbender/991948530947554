#include "hip/hip_runtime.h"

#include <gmp.h>

__global__ void primeSieve(mpz_t n, mpz_t prime1, mpz_t prime2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // thread id
    mpz_t prime;
    mpz_init(prime);
    while (mpz_cmp_ui(n, 0) > 0)
    {
        // find next prime
        mpz_nextprime(prime, n);
 
        if (i % 2 == 0) // assign prime to prime1
            mpz_set(prime1, prime);
        else // assign prime to prime2
            mpz_set(prime2, prime);
 
        // update n
        mpz_sub(n, n, prime);
    }
    mpz_clear(prime);
}
